#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>

#include "kmeans.h"
#include "timer.h"


#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <algorithm>
#include <math.h>
#include <string.h>
#include <fstream>
#include <unistd.h>

using namespace std;

const int MAXD = 1000;//维度
const int MAXN = 1000;//点数
const int MAXC = 50;//Cluster类别

//产生随机标签
void random_labels(thrust::device_vector<int>& labels, int n, int k) {
    thrust::host_vector<int> host_labels(n);
    for(int i = 0; i < n; i++) {
        host_labels[i] = rand() % k;
    }
    labels = host_labels;
}
bool ReadData(thrust::device_vector<double>& data,int m,int d,int index)
{
	thrust::host_vector<double> host_data(m*d);
    std::string filename = "/data/006zzy/files/splitdata_";
    std::string number = std::to_string(index);
    filename += number;
    filename += ".txt";
    ifstream infile;
    infile.open(filename);
    if(!infile) {
    	std::cout <<"read splitdata fail..."<<std::endl;
    	return true;
    }
    for(int i = 0;i < m;i++){
        for(int j = 0;j < d;j++){
            infile >> host_data[i*d + j];
        }
    }
    infile.close();
    data = host_data;
    std::cout << "slave" << index << ": read " <<filename<<" is ok..."<<std::endl;
    //get any 3 points to check read
    /*
    for(int i = 15 ;i < 18;i++){
        	for(int j = 0;j < 20;j++){
        		std::cout <<host_data[i*d + j] << " " ;
        	}
    }
    std::cout << std::endl;
    for(int i = 15 ;i < 18;i++){
    	for(int j = 0;j < 20;j++){
    		std::cout <<data[i*d + j] << " " ;
    	}
    }
    std::cout << std::endl;
    */
    return true;
}
int main(int argc,char* argv[])
{
    //打印参数
	/*
    for(int i = 0;i < argc;i++){
        std::cout << "param " << i << " = "<< argv[i] <<std::endl;
    }*/
    int Cluster_Num = atoi(argv[1]);
    int Point_Num = atoi(argv[2]);
    int Point_Dimension = atoi(argv[3]);
    int Slave_Num = atoi(argv[4]);
    int Slave_Index = atoi(argv[5]);
    int n_gpu = 1;//只使用一个GPU
    int iterations = 1;//只循环1ci
    std::cout << "Use "<< n_gpu << " gpus" << std::endl;

    int Slave_Point_Num = 0;
    double n = (double)Point_Num;
    int every_points = ceil(n/Slave_Num);
    if(!(Point_Num % Slave_Num) || Slave_Index != (Slave_Num - 1)){
        Slave_Point_Num = every_points;
    }
    else if((Point_Num % Slave_Num )&&(Slave_Index == (Slave_Num -1))){
        Slave_Point_Num = Point_Num - (Slave_Index * every_points);
    }
    std::cout <<"This slave has "<<Slave_Point_Num<< "Points..."<<std::endl;

    thrust::device_vector<double> *data[16];
    //这里设置最大是16个GPU，相当于一???6行的指针，每个指针指向一个一维数组，整体相当于一个指向data[16][]的指???分了16组data
    thrust::device_vector<int> *labels[16];//同上
    thrust::device_vector<double> *centroids[16];//同上
    thrust::device_vector<double> *distances[16];//同上
    for (int q = 0; q < n_gpu; q++) {
        hipSetDevice(q);
        data[q] = new thrust::device_vector<double>(Slave_Point_Num/n_gpu*Point_Dimension);//一个GPU管一片数据
        labels[q] = new thrust::device_vector<int>(Slave_Point_Num/n_gpu*Point_Dimension);//一片数据属于哪个类的下标集合
        centroids[q] = new thrust::device_vector<double>(Cluster_Num * Point_Dimension);//存储本片数据得出来的中心???
        distances[q] = new thrust::device_vector<double>(Slave_Point_Num);//不懂为什么创建Point_Num而不是Point_Num/n_gpu
    }
    for (int q = 0; q < n_gpu; q++) {
        ReadData(*data[q],Slave_Point_Num,Point_Dimension,Slave_Index);
        //std::cout <<"ReadData end..."<<std::endl;
        random_labels(*labels[q], Slave_Point_Num, Cluster_Num);//array[n/n_gpu] 但是让他们属于k个标签中的一???    }
        //std::cout <<"random labels end..."<<std::endl;
    }

    kmeans::timer t;
    t.start();
    std::cout <<"start kmeans..."<<std::endl;
    kmeans::kmeans(iterations, Slave_Point_Num, Point_Dimension, Cluster_Num, data, labels, centroids, distances, n_gpu,Slave_Index,false);//执行kmeans,拿到的是所有数
    std::cout <<"end kmeans..."<<std::endl;
    float time = t.stop();
    std::cout << "  Time: " << time/1000.0 << " s" << std::endl;

    for (int q = 0; q < n_gpu; q++) {
       delete(data[q]);
       delete(labels[q]);
       delete(centroids[q]);
    }
}
