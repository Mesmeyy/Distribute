
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "timer.h"

namespace kmeans {

timer::timer() {
    hipEventCreate(&m_start);
    hipEventCreate(&m_stop);
}

timer::~timer() {
    hipEventDestroy(m_start);
    hipEventDestroy(m_stop);
}

void timer::start() {
    hipEventRecord(m_start, 0);
}

float timer::stop() {
    float time;
    hipEventRecord(m_stop, 0);
    hipEventSynchronize(m_stop);
    hipEventElapsedTime(&time, m_start, m_stop);
    return time;
}

}
