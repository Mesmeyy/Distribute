#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/device_allocator.h>
#include <thrust/iterator/counting_iterator.h>

#include "labels.h"
#include <fstream>
using namespace std;

__device__ double myatomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


namespace kmeans {
namespace detail {

__device__ __forceinline__ void update_centroid(int label, int dimension,int d,double accumulator, double* centroids,
                                                int count, int* counts) {
    int index = label * d + dimension;
    double* target = centroids + index;
    myatomicAdd(target, accumulator);
    if (dimension == 0) {
        myatomicAdd((double*)counts + label, count);
    }
}

__global__ void calculate_count(int* ordered_labels,int* counts){
	int dataindex = threadIdx.x + blockIdx.x * blockDim.x;
	int thislabel = ordered_labels[dataindex];
	atomicAdd(counts + thislabel,1);
}

__global__ void calculate_centroids(int n, int d, int k,
                                    double* data,
                                    int* ordered_labels,
                                    int* ordered_indices,
                                    double* centroids){
	int global_id_x = threadIdx.x;
	int global_id_y = threadIdx.y + blockIdx.y * blockDim.y;

	if((global_id_x < d) && (global_id_y < n)){
		int label = ordered_labels[global_id_y];
		int indice = ordered_indices[global_id_y];
		double ademisiondata = data[indice*d + global_id_x];
		double *target = centroids + label * d+global_id_x;
		myatomicAdd(target,ademisiondata);
	}
}
__global__ void scale_centroids(int d, int k, int* counts, double* centroids) {
    int global_id_x = threadIdx.x ;
    int global_id_y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((global_id_x < d) && (global_id_y < k)) {
        int count = counts[global_id_y];
        //To avoid introducing divide by zero errors
        //If a centroid has no weight, we'll do no normalization
        //This will keep its coordinates defined.
        if (count < 1) {
            count = 1;
        }
        double scale = 1.0/double(count);
        centroids[global_id_x + d * global_id_y] *= scale;
    }
}
void Read_Center(int k,int d,thrust::device_vector<double>& centroids){
	thrust::host_vector<double> host_centroids(k*d);
    ifstream infile;
    std::string filename = "/data/006zzy/files/tempcenter.txt";
    infile.open(filename);
    for(int i = 0;i < k;i++){
        for(int j = 0;j < d;j++){
           infile >> host_centroids[i * d + j];
        }
    }
    centroids = host_centroids;
    infile.close();
    std::cout << "slave read tempcenter.txt is ok..." << std::endl;
    /*
    for(int i = 0;i < k;i++)
    	for(int j = 0;j < d;j++){
    		std::cout << centroids[i*d + j] << " ";
    }
    std::cout <<std::endl;
    */

}
void Save_Center(int k,int d,thrust::host_vector<double>& centroids,int index){
	/*
	std::cout <<"h_centroids :"<<std::endl;
	for(int i = 0;i < k;i++){
		for(int j = 0;j < d;j++){
			std::cout << centroids[i*d + j] << " ";
		}
	}
	std::cout<<std::endl;*/
    std::string filename = "/data/006zzy/files/tempdata_";
    std::string number = std::to_string(index);
    filename += number;
    filename += ".txt";
    ofstream outfile;
    outfile.open(filename);
    for(int i = 0;i < k;i++){
        for(int j = 0;j < d;j++){
            outfile << centroids[i * d + j ];
            outfile << " ";
        }
    }
    outfile.close();
}
void find_centroids(int n, int d, int k,
                    thrust::device_vector<double>& data,
                    thrust::device_vector<int>& labels,
                    thrust::device_vector<double>& centroids,
                    thrust::device_vector<int>& range,
                    thrust::device_vector<int>& indices,
                    thrust::device_vector<int>& counts) {
    int dev_num;
    hipGetDevice(&dev_num);
    detail::mymemcpy(indices,range);
    //Bring all labels with the same value together

#if 1
    thrust::sort_by_key(labels.begin(),
                        labels.end(),
                        indices.begin());
#else
    mycub::sort_by_key_int(labels, indices);//wrong!
#endif

    //Initialize centroids to all zeros
    detail::mymemzero(centroids);

    //Initialize counts to all zeros
    detail::mymemzero(counts);

    //Calculate centroids
    int n_threads_x = n;//old:64
    int n_threads_y = 1;//old:16
    //XXX Number of blocks here is hard coded at 30
    //This should be taken care of more thoughtfully.
    //dim3(1,1),old:Dim3(1,30)
    detail::calculate_count<<<dim3(1, 1), dim3(n_threads_x, n_threads_y),
                                  0, hip_stream[dev_num]>>>
        (thrust::raw_pointer_cast(labels.data()),
         thrust::raw_pointer_cast(counts.data()));

    n_threads_x = 512;//old:64
    n_threads_y = 2;//old:16
    //contain 2*128 = 256 points

    detail::calculate_centroids<<<dim3(1, 128), dim3(n_threads_x, n_threads_y),
                                      0, hip_stream[dev_num]>>>
            (n, d, k,
             thrust::raw_pointer_cast(data.data()),
             thrust::raw_pointer_cast(labels.data()),
             thrust::raw_pointer_cast(indices.data()),
             thrust::raw_pointer_cast(centroids.data()));

    //Scale centroids
    n_threads_x = 512;
    n_threads_y = 2;
    //y:k x:d
    detail::scale_centroids<<<dim3(1,128), dim3(n_threads_x, n_threads_y),
                              0, hip_stream[dev_num]>>>
        (d, k,
         thrust::raw_pointer_cast(counts.data()),
         thrust::raw_pointer_cast(centroids.data()));


    /*
    //print counts
    for(int i = 0;i <k;i++){
    	std::cout << counts[i] <<" ";
    }
    std::cout << std::endl;
    //print &new center
    std::cout <<"centroids in find centroids end:"<<std::endl;
    for(int i = 0;i < n;i++){
    	for(int j = 0;j < d;j++){
        	std::cout << centroids[i*d + j] << " ";
        }
    }
    std::cout << std::endl;*/


}

}
}



