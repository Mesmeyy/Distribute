
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include "kmeans.h"
#include "util.h"

namespace kmeans {

int kmeans(int iterations,int n, int d, int k,thrust::device_vector<double>** data,thrust::device_vector<int>** labels,thrust::device_vector<double>** centroids,thrust::device_vector<double>** distances,int n_gpu,int index,bool init_from_labels) {
    thrust::device_vector<double> *data_dots[16];
    thrust::device_vector<double> *centroid_dots[16];
    thrust::device_vector<double> *pairwise_distances[16];
    thrust::device_vector<int> *labels_copy[16];
    thrust::device_vector<int> *range[16];
    thrust::device_vector<int> *indices[16];
    thrust::device_vector<int> *counts[16];

    thrust::host_vector<double> h_centroids( k * d );
    thrust::host_vector<double> h_centroids_tmp( k * d );
    int h_changes[16], *d_changes[16];
    double h_distance_sum[16], *d_distance_sum[16];

    for (int q = 0; q < n_gpu; q++) {
        hipSetDevice(q);
        hipMalloc(&d_changes[q], sizeof(int));
        hipMalloc(&d_distance_sum[q], sizeof(double));
        detail::labels_init();
        data_dots[q] = new thrust::device_vector <double>(n/n_gpu);
        centroid_dots[q] = new thrust::device_vector<double>(n/n_gpu);
        pairwise_distances[q] = new thrust::device_vector<double>(n/n_gpu * k);
        labels_copy[q] = new thrust::device_vector<int>(n/n_gpu * d);//too much
        range[q] = new thrust::device_vector<int>(n/n_gpu);
        counts[q] = new thrust::device_vector<int>(k);
        indices[q] = new thrust::device_vector<int>(n/n_gpu);
        //Create and save "range" for initializing labels
        thrust::copy(thrust::counting_iterator<int>(0),
                     thrust::counting_iterator<int>(n/n_gpu),
                     (*range[q]).begin());
        detail::make_self_dots(n/n_gpu, d, *data[q], *data_dots[q]);
        if (!init_from_labels) {
            //无默认label情况读取中心值(默认情况)
            detail::Read_Center(k,d,*centroids[q]);
        }else{
            //有默认label情况下计算中心值
            detail::find_centroids(n/n_gpu,d,k,*data[q],*labels[q],*centroids[q],*range[q],*indices[q],*counts[q]);
        }
    }
    double prior_distance_sum = 0;
    int i=0;
    for(; i < iterations; i++) {
        for (int q = 0; q < n_gpu; q++) {
            hipSetDevice(q);
            detail::calculate_distances(n/n_gpu, d, k,*data[q], *centroids[q], *data_dots[q],*centroid_dots[q], *pairwise_distances[q]);
            detail::relabel(n/n_gpu, k, *pairwise_distances[q], *labels[q], *distances[q], d_changes[q]);
            detail::mymemcpy(*labels_copy[q], *labels[q]);
            detail::find_centroids(n/n_gpu, d, k, *data[q], *labels[q], *centroids[q], *range[q], *indices[q], *counts[q]);
            detail::mymemcpy(*labels[q], *labels_copy[q]);
            //double d_distance_sum[q] = thrust::reduce(distances[q].begin(), distances[q].end())
            //mycub::sum_reduce(*distances[q], d_distance_sum[q]);
            //Average the centroids from each device
        }
        if (n_gpu >= 1) {
            for (int p = 0; p < k * d; p++) h_centroids[p] = 0.0;
            for (int q = 0; q < n_gpu; q++) {
                hipSetDevice(q);
                detail::mymemcpy(h_centroids_tmp, *centroids[q]);
                detail::streamsync(q);
                for (int p = 0; p < k * d; p++) h_centroids[p] += h_centroids_tmp[p];
            }
            for (int p = 0; p < k * d; p++) h_centroids[p] /= n_gpu;
            detail::Save_Center(k,d,h_centroids,index);
        }
    }
    for (int q = 0; q < n_gpu; q++) {
       hipSetDevice(q);
       hipFree(d_changes[q]);
       detail::labels_close();
       delete(pairwise_distances[q]);
       delete(data_dots[q]);
       delete(centroid_dots[q]);
    }
    return i;
}


}
